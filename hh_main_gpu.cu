#include "hip/hip_runtime.h"
/*
 * hh_main_gpu.cpp
 *
 *  Created on: 20 июля 2016 г.
 *      Author: Pavel Esir
 */

#include <cmath>
#include "hh_main_gpu.h"
#include <cstdio>

#define Cm_    1.0 //  inverse of membrane capacity, 1/pF
#define g_Na  120.0 // nS
#define g_K   36.0
#define g_L   0.3
#define E_K   -77.0
#define E_Na  55.0
#define E_L   -54.4
#define V_peak 25.0

#define NBlockSz 128
#define SBlockSz 512

__device__ float get_random(unsigned int *seed){
    // Park-Miller generator
    // return random number homogeneously distributed in interval [0:1)
    unsigned long a = 16807;
    unsigned long m = 2147483647;
    unsigned long x = (unsigned long) *seed;
    x = (a * x) % m;
    *seed = (unsigned int) x;
    return ((float) x)/m;
}

__device__
float hh_Vm(float V, float n_ch, float m_ch, float h_ch, float I, float h){
    return (-g_K*(V - E_K)*n_ch*n_ch*n_ch*n_ch - g_Na*(V - E_Na)*m_ch*m_ch*m_ch*h_ch - g_L*(V - E_L) + I)*h*Cm_;
}

__device__
float hh_n_ch(float V, float n_ch, float h){
    float temp = 1.0f - exp(-(V + 55.0f)*0.1f);
    if (temp != 0.0f){
        return (.01f*(1.0f - n_ch)*(V + 55.0f)/temp - 0.125f*n_ch*exp(-(V + 65.0f)*0.0125f))*h;
    } else {
//      printf("dividing to zero while calculating n! \n");
//      to understand why it'so, calculate the limit for v/(1 - exp(v/10)) then v tend to 0
        return (0.1f*(1.0f - n_ch)- 0.125f*n_ch*exp(-(V + 65.0f)*0.0125f))*h;
    }
}

__device__
float hh_m_ch(float V, float m_ch, float h){
    float temp = 1.0f - exp(-(V + 40.0f)*0.1f);
    if (temp != 0.0f){
        return (0.1f*(1.0f - m_ch)*(V + 40.0f)/temp - 4.0f*m_ch*exp(-(V + 65.0f)*0.055555556f))*h;
    } else {
//      printf("dividing to zero while calculating  m! \n");
        return ((1.0f - m_ch) - 4.0f*m_ch*exp(-(V + 65.0f)*0.055555556f))*h;
    }
}

__device__
float hh_h_ch(float V, float h_ch, float h){
    return (0.07f*(1.0f - h_ch)*exp(-(V + 65.0f)*0.05f) - h_ch/(1.0f + exp(-(V + 35.0f)*0.1f)))*h;
}

__global__
void integrate_synapses(unsigned int t, unsigned int Ncon, unsigned int Nneur, unsigned int *pre_nidx, unsigned int *post_nidx, float *weight,
                        float *y, unsigned int *delay, unsigned int *num_spike_syn, unsigned int *num_spike_neur, unsigned int *spike_time){
    unsigned int s = blockIdx.x*blockDim.x + threadIdx.x;
    if (s < Ncon){
        // if we processed less spikes than presynaptic neuron generated
        // we need to check whether the new spikes arrive at this moment of time
        if (num_spike_syn[s] < num_spike_neur[pre_nidx[s]]){
            if (spike_time[Nneur*num_spike_syn[s] + pre_nidx[s]] == t - delay[s]){
                atomicAdd(&y[post_nidx[s]], weight[s]);
                num_spike_syn[s]++;
            }
        }
    }
}

__global__
void integrate_neurons(unsigned int t, unsigned int Nneur, float h, float rate, unsigned int *psn_seed, unsigned int *psn_time,
                       float exp_psc, float exp_psc_half, float tau_cor, NeurVars nv, RecordVars rv,
                       unsigned int *num_spike_neur, unsigned int *spike_time, IncSpikes incSpikes){
    unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;
    if (n < Nneur){
        float I_syn_half = (nv.y[n]*h*0.5f + nv.Isyn[n])*exp_psc_half;


        // if where is poisson impulse on neuron
        while (psn_time[n] == t){
            nv.y[n] += nv.weight_p[n];
            // after taking logarithm from uniformly distributed from 0 to 1
            // random number we get exponentially distributed random number
            // for Poisson process time interals between impulses are exponentially distributed
            // sign of right part is negative hence here is "-="
            psn_time[n] += (unsigned int) (-1000.0f*log(get_random(psn_seed + n))/(rate*h));
        }

        while (incSpikes.numProcessed[n] < incSpikes.nums[n] && incSpikes.times[Nneur*incSpikes.numProcessed[n] + n] == t){
            nv.y[n] += incSpikes.weights[Nneur*incSpikes.numProcessed[n] + n];
            incSpikes.numProcessed[n] += 1;
        }
        float V_mem, n_channel, m_channel, h_channel;
        float v1, v2, v3, v4;
        float n1, n2, n3, n4;
        float m1, m2, m3, m4;
        float h1, h2, h3, h4;
        float Inoise_;
        float ns1, ns2, ns3, ns4;

        float dNoise = 0.0f;
    //    float dNoise = sqrtf(2.0f*h*D[n])*hiprand_normal(&state[n]);

        V_mem = nv.V[n];
        n_channel = nv.n[n];
        m_channel = nv.m[n];
        h_channel = nv.h[n];
        Inoise_ = nv.Inoise[n];
        v1 = hh_Vm(nv.V[n], nv.n[n], nv.m[n], nv.h[n], nv.Isyn[n] + nv.Inoise[n] + nv.Ie[n], h);
        n1 = hh_n_ch(nv.V[n], nv.n[n], h);
        m1 = hh_m_ch(nv.V[n], nv.m[n], h);
        h1 = hh_h_ch(nv.V[n], nv.h[n], h);
        ns1 = (-nv.Inoise[n]*h + dNoise)/tau_cor;
        nv.V[n] = V_mem + v1/2.0f;
        nv.n[n] = n_channel + n1/2.0f;
        nv.m[n] = m_channel + m1/2.0f;
        nv.h[n] = h_channel + h1/2.0f;
        nv.Inoise[n] = Inoise_ + ns1/2.0f;

        v2 = hh_Vm(nv.V[n], nv.n[n], nv.m[n], nv.h[n], I_syn_half + nv.Inoise[n] + nv.Ie[n], h);
        n2 = hh_n_ch(nv.V[n], nv.n[n], h);
        m2 = hh_m_ch(nv.V[n], nv.m[n], h);
        h2 = hh_h_ch(nv.V[n], nv.h[n], h);
        ns2 = (-nv.Inoise[n]*h + dNoise)/tau_cor;
        nv.V[n] = V_mem + v2/2.0f;
        nv.n[n] = n_channel + n2/2.0f;
        nv.m[n] = m_channel + m2/2.0f;
        nv.h[n] = h_channel + h2/2.0f;
        nv.Inoise[n] = Inoise_ + ns2/2.0f;


        v3 = hh_Vm(nv.V[n], nv.n[n], nv.m[n], nv.h[n], I_syn_half + nv.Inoise[n] + nv.Ie[n], h);
        n3 = hh_n_ch(nv.V[n], nv.n[n], h);
        m3 = hh_m_ch(nv.V[n], nv.m[n], h);
        h3 = hh_h_ch(nv.V[n], nv.h[n], h);
        ns3 = (-nv.Inoise[n]*h + dNoise)/tau_cor;
        nv.V[n] = V_mem + v3;
        nv.n[n] = n_channel + n3;
        nv.m[n] = m_channel + m3;
        nv.h[n] = h_channel + h3;
        nv.Inoise[n] = Inoise_ + ns3;

        nv.Isyn[n]  = (nv.y[n]*h + nv.Isyn[n])*exp_psc;
        nv.y[n] *= exp_psc;

        v4 = hh_Vm(nv.V[n], nv.n[n], nv.m[n], nv.h[n], nv.Isyn[n] + nv.Inoise[n] + nv.Ie[n], h);
        n4 = hh_n_ch(nv.V[n], nv.n[n], h);
        m4 = hh_m_ch(nv.V[n], nv.m[n], h);
        h4 = hh_h_ch(nv.V[n], nv.h[n], h);
        ns4 = (-nv.Inoise[n]*h + dNoise)/tau_cor;

        nv.V[n] = V_mem + (v1 + 2.0f*(v2 + v3) + v4)/6.0f;
        nv.n[n] = n_channel + (n1 + 2.0f*(n2 + n3) + n4)/6.0f;
        nv.m[n] = m_channel + (m1 + 2.0f*(m2 + m3) + m4)/6.0f;
        nv.h[n] = h_channel + (h1 + 2.0f*(h2 + h3) + h4)/6.0f;
        nv.Inoise[n] = Inoise_ + (ns1 + 2.0f*(ns2 + ns3) + ns4)/6.0f;

        // checking if there's spike on neuron
        if (nv.V[n] > V_peak && V_mem > nv.V[n] && nv.V_last[n] <= V_mem){
            // second condition is necessary in the presence of noise
            if (num_spike_neur[n] == 0 || t - spike_time[Nneur*(num_spike_neur[n] - 1) + n] > 5.0f/h){
                spike_time[Nneur*num_spike_neur[n] + n] = t;
                num_spike_neur[n]++;
            }
        }
        nv.V_last[n] = V_mem;

        if (t % rv.interval == 0){
            rv.V[Nneur*t/rv.interval + n] = nv.V[n];
//            rv.V[Nneur*t/rv.interval + n] = nv.Isyn[n];
        }
    }
}

__global__
void init_noise(unsigned int seed, unsigned int Nneur, float h, float rate, unsigned int *psn_seed, unsigned int *psn_time){
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < Nneur){
        psn_seed[i] = 100000*(seed + i + 1);
        psn_time[i] = 1 + (unsigned int) (-1000.0*log(get_random(psn_seed + i))/(rate*h));
    }
}

__global__
void fillFloatArr(unsigned int size, float *arr, float val){
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < size){
        arr[i] = val;
    }
}

void init_noise_gpu(unsigned int seed, unsigned int Nneur, float h, float rate, unsigned int *psn_seed, unsigned int *psn_time){
    init_noise<<<Nneur/NBlockSz + 1, NBlockSz>>>(seed, Nneur, h, rate, psn_seed, psn_time);
}

void fillFloatArr_gpu(unsigned int size, float *arr, float val){
    fillFloatArr<<<(size + NBlockSz - 1)/NBlockSz, NBlockSz>>>(size, arr, val);
}

void integrate_neurons_gpu(unsigned int t, unsigned int Nneur, float h, float rate, unsigned int *psn_seed, unsigned int *psn_time,
        float exp_psc, float exp_psc_half, float tau_cor, NeurVars nv, RecordVars rv, unsigned int *num_spike_neur, unsigned int *spike_time, IncSpikes incSpikes){
    integrate_neurons<<<(Nneur + NBlockSz - 1)/NBlockSz, NBlockSz>>>(t, Nneur, h, rate, psn_seed, psn_time, exp_psc, exp_psc_half, tau_cor, nv, rv,
            num_spike_neur, spike_time, incSpikes);
}

void integrate_synapses_gpu(unsigned int t, unsigned int Ncon, unsigned int Nneur, unsigned int *pre_nidx, unsigned int *post_nidx, float *weight,
                            float *y, unsigned int *delay, unsigned int *num_spike_syn, unsigned int *num_spike_neur, unsigned int *spike_time){
    integrate_synapses<<<(Ncon + SBlockSz - 1)/SBlockSz, SBlockSz>>>(t, Ncon, Nneur, pre_nidx, post_nidx, weight, y, delay, num_spike_syn, num_spike_neur, spike_time);
}

